#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

__global__ void kernel(int* in, int* out) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = 0;i < 5;i++) {
        out[tid] += in[tid];
    }
}

int main()
{
    const int nStreams = 15;
    const int nBlocks = 65535;
    const int nThreads = 512;
    const int N = 512 * 65535;
    const int size = N * sizeof(int);

    int* host_in;
    int* host_out;

    // 호스트 메모리 할당
    hipHostMalloc((void**)&host_in, size);
    hipHostMalloc((void**)&host_out, size);

    for (int i = 0;i<N;i++) {
        host_in[i] = i;
        host_out[i] = 0;
    }

    int* dev_in, * dev_out;

    // 디바이스 메모리 할당
    hipMalloc((void**)&dev_in, size);
    hipMalloc((void**)&dev_out, size);

    hipMemset(dev_in, 0, size);
    hipMemset(dev_out, 0, size);

    hipEvent_t start, end;
    float time;

    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start, 0);

    // HOST TO DEVICE 데이터 전송
    hipMemcpy(dev_in, host_in, size, hipMemcpyHostToDevice);

    // 덧셈 계산
    kernel << <nBlocks, nThreads >> > (dev_in, dev_out);

    // DEVICE TO HOST 데이터 전송
    hipMemcpy(host_out, dev_out, size, hipMemcpyDeviceToHost);

    hipEventRecord(end, 0); // 측정 종료
    hipEventSynchronize(end);

    hipEventElapsedTime(&time, start, end);

    printf("동기 실행 시간 : %f msec\n", time);

    // ---------------------------------------------------------------------

    for (int i = 0;i < N;i++) {
        host_in[i] = i;
        host_out[i] = 0;
    }

    hipMemset(dev_in, 0, size);
    hipMemset(dev_out, 0, size);

    hipStream_t* streams = (hipStream_t*)malloc(nStreams * sizeof(hipStream_t));

    for (int i = 0;i < nStreams; i++) {
        hipStreamCreate(&(streams[i]));
    }
    
    // 병행 실행 시간 측정
    hipEvent_t streamStart, streamEnd;
    time = 0;

    hipEventCreate(&streamStart);
    hipEventCreate(&streamEnd);

    int offset = 0;

    hipEventRecord(streamStart, 0);

    // HOST TO DEVICE 데이터 전송
    for (int i = 0; i < nStreams; i++) {
        offset = i * N / nStreams;
        hipMemcpyAsync(dev_in + offset, host_in + offset, size, hipMemcpyHostToDevice, streams[i]);
    }

    for (int i = 0; i < nStreams; i++) {
        offset = i * N / nStreams;
        kernel << <nBlocks / nStreams, nThreads, 0, streams[i] >> > (dev_in + offset, dev_out + offset);
    }

    // DEVICE TO HOST 데이터 전송
    for (int i = 0; i < nStreams; i++) {
        offset = i * N / nStreams;
        hipMemcpyAsync(host_out + offset, dev_out + offset, size, hipMemcpyDeviceToHost, streams[i]);
    }

    hipEventRecord(streamEnd, 0);
    hipEventSynchronize(streamEnd);

    hipEventElapsedTime(&time, streamStart, streamEnd);

    printf("스트림 실행 시간 : %f msec\n", time);

    hipEventDestroy(start);
    hipEventDestroy(end);
    hipEventDestroy(streamStart);
    hipEventDestroy(streamEnd);

    for (int i = 0;i < nStreams;i++)
        hipStreamDestroy(streams[i]);

    hipFree(dev_in);
    hipFree(dev_out);

    hipHostFree(host_in);
    hipHostFree(host_out);
    
    return 0;
}


